#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <utility>
#include <algorithm>
#include <unistd.h>
#include <vector>
#include <time.h>
#include "edge.h"
#include "graph.h"
//#include "mapping.h"
#include "GraphCSROpt.h"
#define NGPUS 4

double cTime = 0.0;

// Comparator for sorting edgeList
bool comp(const Edge& lhs, const Edge& rhs)
{
    if(lhs.dest == rhs.dest)
        return (lhs.src < rhs.src);
    else
        return lhs.dest < rhs.dest;
}

Graph_d* copy(Graph* h_graph);
bool handleFile(int argc){
    /* Code for take input graph */
    if(argc < 2){
        std::cout<<"Error : Invalid Arguents\n";
        return false;
    }
    return true;
}

int* EdgesAllowedPerGPU(int number_of_edges){
    int* arr;
    arr = new int[NGPUS];
    int edges_allowed_per_gpu = number_of_edges/NGPUS;
    arr[0] = 0;
    for(int i=1;i<NGPUS;i++){
        arr[i] = edges_allowed_per_gpu;
    }
    arr[NGPUS] = number_of_edges - (NGPUS - 1)*edges_allowed_per_gpu;
    for(int i=1;i<=NGPUS;i++){
        arr[i] = arr[i]+arr[i-1];
    }
    return arr;
}

// void test_graph(Graph graph){
//     std::pair<Graph*,Mapping> subgraph = graph.subGraph(2,7);
//     subgraph.first->print();
//     subgraph.second.print();
// 	Graph_d* d_graph = copyGraphToGPU(subgraph.first);
//     Mapping_d* d_mapping = subgraph.second.copyMappingToGPU();
//     int* allowed_edges = EdgesAllowedPerGPU(graph.e);
// }
//
// std::pair<Graph_d**,Mapping_d**> partition_graph(Graph graph, int* edges_allowed){
//     std::pair<Graph*,Mapping> subgraph[NGPUS];
//     Graph_d* d_graph[NGPUS];
//     Mapping_d* d_mapping[NGPUS];
//     for(int i=0;i<NGPUS;i++){
//         std::cout<<"FOR GPU = "<<i<<std::endl;
//         subgraph[i] = graph.subGraph(edges_allowed[i],edges_allowed[i+1]-1);
//         GraphCSR graphCSR(subgraph[i].first, subgraph[i].second);
//         subgraph[i].first->print();
// 	    subgraph[i].second.print();
//         graphCSR.print();
//         graphCSR.copyCSRToGPU(i);
//         d_graph[i] = copyGraphToGPU(subgraph[i].first);
//         d_mapping[i] = subgraph[i].second.copyMappingToGPU();
//     }
//     std::pair<Graph_d**,Mapping_d**> ret;
//     ret.first = d_graph;
//     ret.second = d_mapping;
//     return ret;
// }

std::vector<std::pair<int,int> > from_to_GPU;
std::vector<int> number_of_vertex(NGPUS);
std::vector<int> number_of_edge(NGPUS);
std::vector<int> number_of_edge_back(NGPUS);
GraphCSROpt_d** partition(Graph *graph){
    //std::cout<<"In Partition\n";
    int* start_point = new int[1];
    start_point[0] = 0;
    int* start_point_back = new int[1];
    start_point_back[0] = 0;
    int v_per_gpu = graph->v/NGPUS;
    if(graph->v%NGPUS != 0)
        v_per_gpu++;
    int total_vertex = graph->v;
    int* allowed_vert = new int[NGPUS+1];
    allowed_vert[0] = 0;
    for(int i=1;i<=NGPUS;i++){
        if(total_vertex >= v_per_gpu){
            allowed_vert[i] = allowed_vert[i-1] + v_per_gpu;
            total_vertex -= v_per_gpu;
        }
        else if(total_vertex > 0 && total_vertex <v_per_gpu){
            allowed_vert[i] = allowed_vert[i-1] + total_vertex;
            total_vertex = 0;
        }
        else
            allowed_vert[i] = allowed_vert[i-1];
    }
    GraphCSROpt_d** partitions = new GraphCSROpt_d*[NGPUS];
    Graph* graph_copy = graph->copy();
    std::sort(graph_copy->edgeList,graph_copy->edgeList + graph_copy->e, comp);
    /*std::cout<<"================\n";
    for(int i=0;i<graph_copy->e;i++)
        std::cout<<graph_copy->edgeList[i].src<<" "<<graph_copy->edgeList[i].dest<<std::endl;
    std::cout<<"================\n";*/
    for(int i=0;i<NGPUS;i++){
        GraphCSROpt temp(graph,allowed_vert[i], allowed_vert[i+1], start_point);
        temp.setForBack(graph_copy,allowed_vert[i], allowed_vert[i+1], start_point_back);
        //temp.print();
        number_of_vertex[i] = temp.v;
        number_of_edge[i] = temp.e;
        number_of_edge_back[i] = temp.e_back;
        partitions[i] = temp.copyToGPU(i);
    }
    return partitions;
}

std::vector<int> inWhichGPU(int vertex){
    std::vector<int> v;
    for(int i=0;i<from_to_GPU.size();i++){
        if(from_to_GPU[i].first <= vertex && from_to_GPU[i].second >= vertex )
            v.push_back(i);
    }
    if(v.size() == 0)
        v.push_back(-1);
    return v;
}

__global__ void coloring_kernel(GraphCSROpt_d* d_graph, int* d_terminate_color){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < d_graph->v && d_graph->d_allowed_v[tid] > 0){
        for(int i=d_graph->d_a[tid];i<d_graph->d_a[tid+1];i++){
            if(d_graph->d_b[i] >= 0 && d_graph->d_b[i] < d_graph->v && d_graph->d_allowed_v[d_graph->d_b[i]] > 0){
                // This case is for if vertex is present in gpu
                d_graph->d_old[tid] = atomicMax(&(d_graph->d_color_v[d_graph->d_b[i]]), d_graph->d_color_v[tid]);
                atomicCAS(&(d_graph->d_old[tid]), d_graph->d_color_v[d_graph->d_b[i]] , -10);
                if(d_graph->d_old[tid] != -10)
                    atomicCAS(&d_terminate_color[0], 1 , 0);
            }
            else /*if(d_graph->d_b[i] < 0 && d_graph->d_b[i] >= d_graph->v)*/{
                // If Vertex is not present in GPU
                atomicMax(&(d_graph->d_color_e[i]), d_graph->d_color_v[tid]);
            }
        }
    }
    else if(tid < d_graph->v && d_graph->d_allowed_v[tid] <= 0){
        for(int i=d_graph->d_a[tid];i<d_graph->d_a[tid+1];i++){
            d_graph->d_color_e[i] = -1;
        }
    }
}

__global__ void kernel_BWD(GraphCSROpt_d* d_graph, int* d_terminate_BWD){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < d_graph->v && d_graph->d_allowed_v[tid] > 0 && d_graph->d_pivots[tid] >= 0){
        for(int i=d_graph->d_c[tid];i<d_graph->d_c[tid+1];i++){
            int index = d_graph->d_d[i];
            if(index >= 0 && index < d_graph->v && d_graph->d_allowed_v[index] > 0){
                if(d_graph->d_color_v[tid] == d_graph->d_color_v[index]){
                    //printf("IN KERNEL BWD\n");
                d_graph->d_old[tid] = atomicCAS(&(d_graph->d_pivots[index]), -1, d_graph->d_pivots[tid]);
                atomicCAS(&(d_graph->d_old[tid]), d_graph->d_pivots[index], -5);
                if(d_graph->d_old[tid] != -5)
                    atomicCAS(&d_terminate_BWD[0], 1 , 0);
                }
            }
            else{
                d_graph->d_bwd_e[i] = d_graph->d_pivots[tid];
                d_graph->d_color_bwd_e[i] = d_graph->d_color_v[tid];
            }
        }
    }
    else if(tid < d_graph->v && d_graph->d_allowed_v[tid] > 0 && d_graph->d_pivots[tid] < 0){
        for(int i=d_graph->d_c[tid];i<d_graph->d_c[tid+1];i++){
            d_graph->d_bwd_e[i] = d_graph->d_pivots[tid];
            d_graph->d_color_bwd_e[i] = -67;
        }
    }
    else if(tid < d_graph->v && d_graph->d_allowed_v[tid] <= 0){
        for(int i=d_graph->d_c[tid];i<d_graph->d_c[tid+1];i++){
            d_graph->d_bwd_e[i] = -1;
            d_graph->d_color_bwd_e[i] = -67;
        }
    }
}

__global__ void print_coloring(GraphCSROpt_d* d_graph) {
    printf("[DEBUG] in COLORING kernel -->\n");
    printf("====================\n");
    for(int i=0;i<d_graph->v;i++){
        printf("%d  --> %d\n",i+d_graph->offset,d_graph->d_color_v[i]);
    }
    printf("====================\n");
}

__global__ void print_BWD(GraphCSROpt_d* d_graph) {
    printf("[DEBUG] in BWD kernel -->\n");
    printf("====================\n");
    for(int i=0;i<d_graph->v;i++){
        printf("%d  --> %d\n",i+d_graph->offset,d_graph->d_pivots[i]);
    }
    printf("====================\n");
}


__global__ void update_kernel(GraphCSROpt_d* d_graph, int* d_v, int* d_e, int* d_offset,int* d_diff_color_e, int* d_dest_vert, int* d_terminate_update){
    // write your update kernel here
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    /*if(tid == 0){
        printf("Degub [kernel] [update] :-> \n");
        printf("d_v = %d, d_e = %d, d_offset = %d\n", d_v[0], d_e[0], d_offset[0]);
    }*/
    if(tid < d_e[0] && d_diff_color_e[tid] >= 0){
        int index = d_dest_vert[tid] + d_offset[0]-d_graph->offset;
        if(index < d_graph->v && index >= 0 && d_graph->d_allowed_v[index] >= 0){
            d_graph->d_old[tid] = atomicMax(&(d_graph->d_color_v[index]), d_diff_color_e[tid]);
            atomicCAS(&(d_graph->d_old[tid]), d_graph->d_color_v[index] , -900);  // Not sure correct
            if(d_graph->d_old[tid] != -900){
                atomicCAS(&d_terminate_update[0], 1, 0);
            }
        }
    }
}

__global__ void update_kernel_BWD(GraphCSROpt_d* d_graph, int* d_v, int* d_e, int* d_offset,int* d_diff_color_e, int* d_dest_vert, int* d_color_bwd_e,int* d_terminate_update){
    // write your update kernel here
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    /*if(tid == 0){
        printf("Degub [kernel] [update] :-> \n");
        printf("d_v = %d, d_e = %d, d_offset = %d\n", d_v[0], d_e[0], d_offset[0]);
    }*/
    if(tid < d_e[0] && d_diff_color_e[tid] >= 0){
        int index = d_dest_vert[tid] + d_offset[0]-d_graph->offset;
        if(index < d_graph->v && index >= 0 && d_graph->d_allowed_v[index] >= 0 && d_diff_color_e[tid] == d_graph->d_color_v[index]){
		//printf("%d %d\n",d_color_bwd_e[tid], d_graph->d_color_v[index]);
            //d_graph->d_old[tid] = atomicMax(&(d_graph->d_color_v[index]), d_diff_color_e[tid]);
            d_graph->d_old[tid] = atomicCAS(&(d_graph->d_pivots[index]), -1, d_diff_color_e[tid]);
            atomicCAS(&(d_graph->d_old[tid]), d_graph->d_pivots[index] , -1000);  // Not sure correct
            if(d_graph->d_old[tid] != -1000){
                atomicCAS(&d_terminate_update[0], 1, 0);
            }
        }
    }
}
/*__global__ void print_check(int* d_v){
    printf("[Debug] [In Kernel Check] : d_v = %d\n",d_v[0]);
}*/

bool  updateToAnotherGPUS(GraphCSROpt_d** d_part, int** terminate_address){
    int number_of_blocks_e[NGPUS];
    int number_of_threads_per_block_e[NGPUS];
    for(int i=0;i<NGPUS;i++){
        number_of_blocks_e[i] = ceil((number_of_edge[i]*1.0)/512.0);
        number_of_threads_per_block_e[i] = ceil(number_of_edge[i]/number_of_blocks_e[i])+1;
    }
    int* arr_v = new int[NGPUS];
    int* arr_e = new int[NGPUS];
    int** arr_color_e = new int*[NGPUS];
    int** arr_dest_vert = new int*[NGPUS];
    int* arr_offset = new int[NGPUS];
    clock_t tStart = clock();
    for(int i=0;i<NGPUS;i++){
        // Now gpu i will update to another GPUS
        // First copy content of i th GPU to CPU
        int v;
        int e;
        int* color_e;
        int* dest_vert;
        int offset;
        int* temp_addr;
        hipSetDevice(i);
        //std::cout<<"Debug [Loop 1] <Device> "<<i<<std::endl;
        hipMemcpy(&(offset), &(d_part[i]->offset), sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&(v), &(d_part[i]->v), sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&(e), &(d_part[i])->e, sizeof(int), hipMemcpyDeviceToHost);
        color_e = (int *)malloc(e*sizeof(int));
        dest_vert = (int *)malloc(e*sizeof(int));
        //std::cout<<"Debug [1] : e = "<<e<<std::endl;
        hipMemcpy(&temp_addr,&d_part[i]->d_b, sizeof(int *), hipMemcpyDeviceToHost);
        hipMemcpy(dest_vert, temp_addr, e*sizeof(int), hipMemcpyDeviceToHost);
        //std::cout<<"Deug [2] : dest_vert[1] = "<<dest_vert[1]<<std::endl;
        hipMemcpy(&temp_addr,&d_part[i]->d_color_e, sizeof(int *), hipMemcpyDeviceToHost);
        hipMemcpy(color_e,temp_addr, e*sizeof(int), hipMemcpyDeviceToHost);
        arr_v[i] = v;
        arr_e[i] = e;
        arr_offset[i] = offset;
        arr_color_e[i] = color_e;
        arr_dest_vert[i] = dest_vert;
        //std::cout<<"Deug [2] : v  = "<<arr_v[i]<<std::endl;
    }
    clock_t tEnd = clock();
    cTime += (double)(tEnd - tStart)/CLOCKS_PER_SEC;
    //std::cout<<"Debug [1] [v]: "<<arr_v[0]<<" "<<arr_v[1]<<std::endl;
    // Now we have all we need in the CPU
    // Now we have to copy these on different GPUS
    for(int i=0;i<NGPUS;i++){
        for(int j=0;j<NGPUS;j++){
            if(i == j)
                continue;
            //std::cout<<"[Debug] [Touple] :- <"<<i<<","<<j<<">"<<std::endl;
            hipSetDevice(j);
            clock_t tStart = clock();
            int* d_v;
            int* d_e;
            int* d_diff_color_e;
            int* d_dest_vert;
            int* d_offset;
            hipMalloc((void **)&d_v, sizeof(int));
            hipMalloc((void **)&d_e, sizeof(int));
            hipMalloc((void **)&d_offset, sizeof(int));
            hipMemcpy(&d_v, &arr_v[i],sizeof(int), hipMemcpyHostToDevice);
            hipMalloc((void **)&d_diff_color_e, arr_e[i]*sizeof(int));
            hipMalloc((void **)&d_dest_vert,arr_e[i]*sizeof(int));
            hipMemcpy(d_v, (arr_v+i), sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_e, (arr_e+i),sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_offset, (arr_offset + i), sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_diff_color_e,arr_color_e[i], arr_e[i]*sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_dest_vert, arr_dest_vert[i], arr_e[i]*sizeof(int), hipMemcpyHostToDevice);
            clock_t tEnd = clock();
            cTime += (double)(tEnd - tStart)/CLOCKS_PER_SEC;
            //std::cout<<"Debug[1] : "<<offset<<std::endl;
            update_kernel<<<number_of_blocks_e[i], number_of_threads_per_block_e[i]>>>(d_part[j], d_v, d_e, d_offset, d_diff_color_e, d_dest_vert,terminate_address[j]);
            //hipDeviceSynchronize();
            hipFree(d_v);
            hipFree(d_e);
            hipFree(d_diff_color_e);
            hipFree(d_offset);
            hipFree(d_dest_vert);
            int h_terminate;
            hipMemcpy(&h_terminate, terminate_address[j], sizeof(int), hipMemcpyDeviceToHost);
            if(h_terminate == 0)
                return false;
        }
    }
    return true;
}

bool  updateBWDToAnotherGPUS(GraphCSROpt_d** d_part, int** terminate_address){
    int number_of_blocks_e_back[NGPUS];
    int number_of_threads_per_block_e_back[NGPUS];
    for(int i=0;i<NGPUS;i++){
        number_of_blocks_e_back[i] = ceil((number_of_edge_back[i]*1.0)/512.0);
        number_of_threads_per_block_e_back[i] = ceil(number_of_edge_back[i]/number_of_blocks_e_back[i])+1;
    }
    int* arr_v = new int[NGPUS];
    int* arr_e = new int[NGPUS];
    int** arr_color_e = new int*[NGPUS];
    int** arr_dest_vert = new int*[NGPUS];
    int** arr_color_bwd_e = new int*[NGPUS];
    int* arr_offset = new int[NGPUS];
    clock_t tStart = clock();
    for(int i=0;i<NGPUS;i++){
        // Now gpu i will update to another GPUS
        // First copy content of i th GPU to CPU
        int v;
        int e;
        int* color_e;
        int* dest_vert;
        int offset;
        int* temp_addr;
        int* color_bwd_e;
        hipSetDevice(i);
        //std::cout<<"Debug [Loop 1] <Device> "<<i<<std::endl;
        hipMemcpy(&(offset), &(d_part[i]->offset), sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&(v), &(d_part[i]->v), sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&(e), &(d_part[i])->e_back, sizeof(int), hipMemcpyDeviceToHost);
        color_e = (int *)malloc(e*sizeof(int));
        dest_vert = (int *)malloc(e*sizeof(int));
	color_bwd_e = (int *)malloc(e*sizeof(int));
        //std::cout<<"Debug [1] : e = "<<e<<std::endl;
        hipMemcpy(&temp_addr,&d_part[i]->d_d, sizeof(int *), hipMemcpyDeviceToHost);
        hipMemcpy(dest_vert, temp_addr, e*sizeof(int), hipMemcpyDeviceToHost);
        //std::cout<<"Deug [2] : dest_vert[1] = "<<dest_vert[1]<<std::endl;
        hipMemcpy(&temp_addr,&d_part[i]->d_bwd_e, sizeof(int *), hipMemcpyDeviceToHost);
        hipMemcpy(color_e,temp_addr, e*sizeof(int), hipMemcpyDeviceToHost);

        hipMemcpy(&temp_addr,&d_part[i]->d_color_bwd_e, sizeof(int *), hipMemcpyDeviceToHost);
        hipMemcpy(color_bwd_e,temp_addr, e*sizeof(int), hipMemcpyDeviceToHost);
        arr_v[i] = v;
        arr_e[i] = e;
        arr_offset[i] = offset;
        arr_color_e[i] = color_e;
        arr_dest_vert[i] = dest_vert;
        arr_color_bwd_e[i] = color_bwd_e;
        //std::cout<<"Deug [2] : v  = "<<arr_v[i]<<std::endl;
    }
    clock_t tEnd = clock();
    cTime += (double)(tEnd - tStart)/CLOCKS_PER_SEC;
    //std::cout<<"Debug [1] [v]: "<<arr_v[0]<<" "<<arr_v[1]<<std::endl;
    // Now we have all we need in the CPU
    // Now we have to copy these on different GPUS
    for(int i=0;i<NGPUS;i++){
        for(int j=0;j<NGPUS;j++){
            if(i == j)
                continue;
            //std::cout<<"[Debug] [Touple] :- <"<<i<<","<<j<<">"<<std::endl;
            hipSetDevice(j);
            clock_t tStart = clock();
            int* d_v;
            int* d_e;
            int* d_diff_color_e;
            int* d_dest_vert;
            int* d_offset;
            int* d_color_bwd_e;
            hipMalloc((void **)&d_v, sizeof(int));
            hipMalloc((void **)&d_e, sizeof(int));
            hipMalloc((void **)&d_offset, sizeof(int));
            hipMemcpy(&d_v, &arr_v[i],sizeof(int), hipMemcpyHostToDevice);
            hipMalloc((void **)&d_diff_color_e, arr_e[i]*sizeof(int));
            hipMalloc((void **)&d_dest_vert,arr_e[i]*sizeof(int));
            hipMalloc((void **)&d_color_bwd_e,arr_e[i]*sizeof(int));
            hipMemcpy(d_v, (arr_v+i), sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_e, (arr_e+i),sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_offset, (arr_offset + i), sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_diff_color_e,arr_color_e[i], arr_e[i]*sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_dest_vert, arr_dest_vert[i], arr_e[i]*sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_color_bwd_e, arr_color_bwd_e[i], arr_e[i]*sizeof(int), hipMemcpyHostToDevice);
            clock_t tEnd = clock();
            cTime += (double)(tEnd - tStart)/CLOCKS_PER_SEC;
            //std::cout<<"Debug[1] : "<<offset<<std::endl;
            update_kernel_BWD<<<number_of_blocks_e_back[i], number_of_threads_per_block_e_back[i]>>>(d_part[j], d_v, d_e, d_offset, d_diff_color_e, d_dest_vert,d_color_bwd_e,terminate_address[j]);
            //hipDeviceSynchronize();
            hipFree(d_v);
            hipFree(d_e);
            hipFree(d_diff_color_e);
            hipFree(d_offset);
            hipFree(d_dest_vert);
            int h_terminate;
            hipMemcpy(&h_terminate, terminate_address[j], sizeof(int), hipMemcpyDeviceToHost);
            if(h_terminate == 0)
                return false;
        }
	//hipDeviceSynchronize();
    }
    return true;
}

//__global__ void print_terminate_address(int*)
void coloring(GraphCSROpt_d** d_part){
    int number_of_blocks[NGPUS];
    int number_of_thread_per_block[NGPUS];

    for(int i=0;i<NGPUS;i++){
        number_of_blocks[i] = ceil((number_of_vertex[i]*1.0)/512.0);
        number_of_thread_per_block[i] = ceil(number_of_vertex[i]/number_of_blocks[i])+1;
    }
    int** terminate_address = new int*[NGPUS];
    for(int i=0;i<NGPUS;i++){
        hipSetDevice(i);
        int* d_terminate_color;
        hipMalloc((void **)&d_terminate_color, sizeof(int));
        terminate_address[i] = d_terminate_color;
    }
    int* h_terminate_color = new int[NGPUS];
    for(int i=0;i<NGPUS;i++)
        h_terminate_color[i] = 0;
    bool terminate = false;
    int temp = 1;
    while(!terminate){
        // Color inside the gpus only
        while(!terminate){
            terminate = true;
            for(int i=0;i<NGPUS;i++){
                hipSetDevice(i);
                hipMemcpy(terminate_address[i], &temp, sizeof(int), hipMemcpyHostToDevice);
                coloring_kernel<<<number_of_blocks[i],number_of_thread_per_block[i]>>>(d_part[i],terminate_address[i]);
                hipMemcpy(&h_terminate_color[i], terminate_address[i], sizeof(int), hipMemcpyDeviceToHost);
                if(h_terminate_color[i] == 0)
                    terminate = false;
            }
        }
        for(int i=0;i<NGPUS;i++){
            hipSetDevice(i);
            hipMemcpy(terminate_address[i], &temp, sizeof(int), hipMemcpyHostToDevice);
        }
        terminate = updateToAnotherGPUS(d_part,terminate_address);
    }
    for(int i=0;i<NGPUS;i++){
        //std::cout<<"Updating["<<i<<"] .. \n";
        hipSetDevice(i);
        //print_coloring<<<1,1>>>(d_part[i]);
        hipDeviceSynchronize();
    }

}

__global__ void kernel_pivots(GraphCSROpt_d* d_graph, int* d_total){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < d_graph->v){
        if(d_graph->d_allowed_v[tid] > 0 && d_graph->offset+tid == d_graph->d_color_v[tid]){
            d_graph->d_pivots[tid] = d_graph->d_color_v[tid];
	//	printf("PIVOT IS = %d\n",tid+d_graph->offset);
            atomicAdd(&d_total[0],1);
        }
    }
}

int selectPivots(GraphCSROpt_d** d_part){
    int number_of_blocks[NGPUS];
    int number_of_thread_per_block[NGPUS];

    for(int i=0;i<NGPUS;i++){
        number_of_blocks[i] = ceil((number_of_vertex[i]*1.0)/512.0);
        number_of_thread_per_block[i] = ceil(number_of_vertex[i]/number_of_blocks[i])+1;
    }
    int* h_temp = new int[1];
    h_temp[0] = 0;
    int* h_total = new int[NGPUS];
    for(int i=0;i<NGPUS;i++){
        hipSetDevice(i);
        int* d_total;
        hipMalloc((void **)&d_total,sizeof(int));
        hipMemcpy(d_total,h_temp, sizeof(int), hipMemcpyHostToDevice);
        kernel_pivots<<<number_of_blocks[i], number_of_thread_per_block[i]>>>(d_part[i], d_total);
        hipMemcpy(h_total+i, d_total, sizeof(int), hipMemcpyDeviceToHost);
    }
    int total = 0;
    for(int i=0;i<NGPUS;i++)
        total += h_total[i];
    return total;
}


void backwardClosure(GraphCSROpt_d** d_part){
    int number_of_blocks[NGPUS];
    int number_of_thread_per_block[NGPUS];

    for(int i=0;i<NGPUS;i++){
        number_of_blocks[i] = ceil((number_of_vertex[i]*1.0)/512.0);
        number_of_thread_per_block[i] = ceil(number_of_vertex[i]/number_of_blocks[i])+1;
    }
    int** terminate_address = new int*[NGPUS];
    for(int i=0;i<NGPUS;i++){
        hipSetDevice(i);
        int* d_terminate_color;
        hipMalloc((void **)&d_terminate_color, sizeof(int));
        terminate_address[i] = d_terminate_color;
    }

    int* h_terminate_color = new int[NGPUS];
    for(int i=0;i<NGPUS;i++)
        h_terminate_color[i] = 0;
    bool terminate = false;
    int temp = 1;
    while(!terminate){
        while(!terminate){
            terminate = true;
            for(int i=0;i<NGPUS;i++){
                hipSetDevice(i);
                hipMemcpy(terminate_address[i], &temp, sizeof(int), hipMemcpyHostToDevice);
                kernel_BWD<<<number_of_blocks[i],number_of_thread_per_block[i]>>>(d_part[i],terminate_address[i]);
                //print_BWD<<<1,1>>>(d_part[i]);
                //hipDeviceSynchronize();
                hipMemcpy(&h_terminate_color[i], terminate_address[i], sizeof(int), hipMemcpyDeviceToHost);
                if(h_terminate_color[i] == 0)
                    terminate = false;
            }
        }
        for(int i=0;i<NGPUS;i++){
            hipSetDevice(i);
            hipMemcpy(terminate_address[i], &temp, sizeof(int), hipMemcpyHostToDevice);
        }
        //std::cout<<"CAME to update"<<std::endl;
        terminate = updateBWDToAnotherGPUS(d_part,terminate_address);
    }
    for(int i=0;i<NGPUS;i++){
        //std::cout<<"Updating["<<i<<"] .. \n";
        hipSetDevice(i);
        //print_BWD<<<1,1>>>(d_part[i]);
        hipDeviceSynchronize();
    }

}

__global__ void kernel_scc(GraphCSROpt_d* d_graph, int* d_total){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < d_graph->v && d_graph->d_allowed_v[tid] > 0){
        if(d_graph->d_color_v[tid] == d_graph->d_pivots[tid]){
            d_graph->d_allowed_v[tid] = -1;
            atomicAdd(&d_total[0],1);
        }
    }
}

int remove_scc(GraphCSROpt_d** d_part){
    int number_of_blocks[NGPUS];
    int number_of_thread_per_block[NGPUS];

    for(int i=0;i<NGPUS;i++){
        number_of_blocks[i] = ceil((number_of_vertex[i]*1.0)/512.0);
        number_of_thread_per_block[i] = ceil(number_of_vertex[i]/number_of_blocks[i])+1;
    }
    int total = 0;
    int* h_temp = new int[1];
    h_temp[0] = 0;
    int* h_total = new int[NGPUS];
    for(int i=0;i<NGPUS;i++){
        hipSetDevice(i);
        int* d_total;
        hipMalloc((void **)&d_total, sizeof(int));
        hipMemcpy(d_total,h_temp, sizeof(int), hipMemcpyHostToDevice);
        kernel_scc<<<number_of_blocks[i], number_of_thread_per_block[i]>>>(d_part[i], d_total);
        hipMemcpy(h_total+i, d_total,sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<NGPUS;i++)
        total += h_total[i];
    return total;
}

__global__ void kernel_Init_Color(GraphCSROpt_d* d_graph){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < d_graph->v && d_graph->d_allowed_v[tid] > 0){
        d_graph->d_color_v[tid] = tid + d_graph->offset;
    }
    else if(tid < d_graph->v){
        d_graph->d_color_v[tid] = -1;
    }
}

__global__ void kernel_init_Color_e(GraphCSROpt_d* d_graph){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < d_graph->e){
        d_graph->d_color_e[tid] = d_graph->d_b[tid] + d_graph->offset;
    }
}

void init_coloring(GraphCSROpt_d** d_part){
    int number_of_blocks[NGPUS];
    int number_of_thread_per_block[NGPUS];
    int number_of_blocks_e[NGPUS];
    int number_of_thread_per_block_e[NGPUS];

    for(int i=0;i<NGPUS;i++){
        number_of_blocks[i] = ceil((number_of_vertex[i]*1.0)/512.0);
        number_of_thread_per_block[i] = ceil(number_of_vertex[i]/number_of_blocks[i])+1;
        number_of_blocks_e[i] = ceil((number_of_edge[i]*1.0)/512.0);
        number_of_thread_per_block_e[i] = ceil(number_of_edge[i]/number_of_blocks_e[i])+1;
    }
    for(int i=0;i<NGPUS;i++){
        hipSetDevice(i);
        kernel_Init_Color<<<number_of_blocks[i], number_of_thread_per_block[i]>>>(d_part[i]);
        kernel_init_Color_e<<<number_of_blocks_e[i], number_of_thread_per_block_e[i]>>>(d_part[i]);
        //print_coloring<<<1,1>>>(d_part[i]);
        hipDeviceSynchronize();
    }
}

int main(int argc, char const *argv[]){
    //const unsigned long long MEGABYTE = 500*1024 * 1024;
    //hipDeviceSetLimit(hipLimitPrintfFifoSize,MEGABYTE);
    bool hfile = handleFile(argc);
    if(hfile){
        /* Reading from file that is given by CLA */
	std::cout<<"Reading Graph .."<<std::endl;
        Graph graph(argv[1]);
        //std::cout<<"CAME HERE\n";
        //test_graph(graph);
        //int* allowed_edges = EdgesAllowedPerGPU(graph.e);
        //std::pair<Graph_d**,Mapping_d**> part = partition_graph(graph,allowed_edges);
	std::cout<<"Partiton Graph .."<<std::endl;
        GraphCSROpt_d** d_part = partition(&graph);
        //std::cout<<"CAME HERE\n";
        /*for(int i=0;i<from_to_GPU.size();i++){
            std::cout<<"GPU ID = "<<i<<std::endl;
            std::cout<<"FROM = "<<from_to_GPU[i].first<<" TO = "<<from_to_GPU[i].second<<std::endl;
        }*/
        int total_vert = graph.v;
        int total_scc = 0;
	clock_t tStart = clock();
	std::cout<<"Partiton Done .."<<std::endl;
        while(total_vert > 0){
        //for(int i=0;i<1;i++){
	//	std::cout<<"COLORING -->"<<std::endl;
            coloring(d_part);
	//	std::cout<<"PIVOT SELECTION -->"<<std::endl;
            total_scc += selectPivots(d_part);
	//	std::cout<<"BACKWARD CLOSURE -->"<<std::endl;
		//sleep(3);
            backwardClosure(d_part);
	//	std::cout<<"REMOVING SCCs -->"<<std::endl;
            total_vert -= remove_scc(d_part);
	//	std::cout<<"INIT COLORING -->"<<std::endl;
            init_coloring(d_part);
        //    std::cout<<total_vert<<std::endl;
            //sleep(3);
        }
	clock_t tEnd = clock();
        std::cout<<"Total SCC = "<<total_scc<<std::endl;
	printf("Time taken: %.2fs\n", (double)(tEnd - tStart)/CLOCKS_PER_SEC);
	printf("Communication Time %.2f\n", cTime);
        //std::cout<<remove_scc(d_part)<<std::endl;
    }
    return 0;
}
