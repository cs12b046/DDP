
#include <hip/hip_runtime.h>
#include <iostream>
#include <malloc.h>

using namespace std;
__global__ void add(int* d_a, int* d_b, int* d_c, int* d_limit){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if(tid < 1000){
		d_c[tid] = d_a[tid] + d_b[tid];
	}
}
int main(){
	int size = 2000; // size of an array
	int ngpus = 2;

	/* Device memory pointer for storing array*/
	int *d_a[2], *d_b[2], *d_c[2];
	const int Ns[2] = {size/2, size - size/2};

	/* memory allocation for limit */
	int* h_limit;
	int* d_limit;
	h_limit = (int *)malloc(sizeof(int));
	hipMalloc((void **)&d_limit, sizeof(int));
	
	/* Host memory for storing array */
	int h_a[size];
	int h_b[size];
	for(int i=0;i<size;i++){
		h_a[i] = i+1;
		h_b[i] = i+2;
	}
	/*int* h_c[ngpus];
	for(int dev=0; dev < ngpus; dev++){
		h_c[dev] = (int *)malloc(Ns[dev]*sizeof(int));		
	}*/
	
	int* h_c;
	h_c = (int *)malloc(size*sizeof(int));

	/* allocate memory on gpus */
	for(int dev=0; dev< ngpus ;dev++){
		hipSetDevice(dev);
		hipMalloc((void **)&d_a[dev], Ns[dev]*sizeof(int));
		hipMalloc((void **)&d_b[dev], Ns[dev]*sizeof(int));
		hipMalloc((void **)&d_c[dev], Ns[dev]*sizeof(int));
	}
	
	/* Copy the host array to gpus */
	for(int dev=0,pos=0; dev < ngpus; pos+= Ns[dev], dev++){
		hipSetDevice(dev);
		hipMemcpy(d_a[dev], h_a+pos, Ns[dev]*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_b[dev], h_b+pos, Ns[dev]*sizeof(int), hipMemcpyHostToDevice);
	}
	
	/* Compute addition */
	for(int dev=0; dev< ngpus; dev++){
		//h_limit[0] = Ns[dev];
		hipSetDevice(dev);
		h_limit[0] = Ns[dev];
		hipMemcpy(d_limit, h_limit, sizeof(int), hipMemcpyHostToDevice);
		add<<<1,Ns[dev]>>>(d_a[dev],d_b[dev], d_c[dev], d_limit);
		/*cudaMemcpy(h_c[dev], d_c[dev], Ns[dev]*sizeof(int), cudaMemcpyDeviceToHost);
		for(int i=0;i<Ns[dev];i++){
			if(i%100 == 0)
				cout<<h_c[dev][i]<<endl;
		}*/
	}
	
	for(int dev=0, pos=0; dev < ngpus; pos += Ns[dev], dev++){
		hipSetDevice(dev);
		hipMemcpy(h_c+pos, d_c[dev], Ns[dev]*sizeof(int), hipMemcpyDeviceToHost);
	}

	/* Print Part */
	for(int i=0;i<size;i++){
		if(i%100 == 0)
			cout<<"h_c["<<i<<"] = "<<h_c[i]<<endl;
	}
}

