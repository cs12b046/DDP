
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* d_a, int* d_b, int* d_c){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if(tid < 2000){
		d_c[tid] = d_a[tid] + d_b[tid];
	}
}

int main(int argc, char* argv[]){
	hipSetDevice(1);
	return 0;
}

