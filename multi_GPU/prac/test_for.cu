
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void print(){
	printf("KYU NHI CHAL RHAA\n");
}

int main(){
	int n = 3;
	int x[n];
	x[0] = 0;
	x[1] = 1;
	x[2] = 2;
	for(int i=0;i<n;++i){
		for(int j=0;j<n;++j){
			if(i != j){
				std::cout<<i<<" "<<j<<std::endl;
				hipSetDevice(j);
                        	int* d_v;
                       		hipMalloc((void **)&d_v, sizeof(int));
                        	hipMemcpy(d_v, &x[i], sizeof(int), hipMemcpyHostToDevice);
				print<<<1,1>>>();
				hipDeviceSynchronize();
			}
			//std::cout<<i<<" "<<j<<std::endl;
			/*cudaSetDevice(j);
			int* d_v;
			cudaMalloc((void **)&d_v, sizeof(int));
			cudaMemcpy(d_v, &x[i], sizeof(int), cudaMemcpyHostToDevice);*/
		}
	}
	for(int i=0;i<n;i++){
		hipSetDevice(i);
		for(int j=0;j<1000;j++){
			//std::cout<<j<<std::endl;
			print<<<1,1>>>();
                	hipDeviceSynchronize();
		}
		//print<<<1,1>>>();
		//cudaDeviceSynchronize();
	}
	for(int i=0;i<n;i++){
                hipSetDevice(i);
                for(int j=0;j<1000;j++){
                        //std::cout<<j<<std::endl;
                        print<<<1,1>>>();
                        hipDeviceSynchronize();
                }
                //print<<<1,1>>>();
                //cudaDeviceSynchronize();
        }

}
