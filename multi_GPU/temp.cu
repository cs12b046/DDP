#include "hip/hip_runtime.h"
/* Make Sure you select correct pivot
 * required int* d_pivots of size v
 * Make sure you make a global variable of total_number_of_vertex
 * Create a array global that finds out prefix_sum of v*/

__global__ void kernel_pivots(GraphCSROpt_d* d_graph){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < d_graph->v){
        if(d_graph->offset+tid == d_graph->d_color_v[tid]){
            d_graph->d_pivots[tid] = d_graph->d_color_v[tid];
        }
    }
}

int* selectPivots(GraphCSROpt_d** d_part, int* number_of_blocks, int* number_of_thread_per_block){
    for(int i=0;i<NGPUS;i++){
        hipSetDevice(i);
        kernel_pivots<<<number_of_blocks[i], number_of_thread_per_block[i]>>>(d_part[i]);
    }
    int* h_pivots = new int[total_number_of_vertex];
    for(int i=0;i<NGPUS;i++){
        hipSetDevice(i);
        hipMemcpy((h_pivots+prefix_sum[i]),d_graph->d_pivots, d_part[i]->v*sizeof(int), hipMemcpyDeviceToHost);
    }
    return h_pivots;
}

int* startPointForCopy(Bwd** h_part){
    int* arr = new int[NGPUS];
    arr[0] = 0;
    for(i=1;i<NGPUS;i++){
        arr[i] = arr[i-1]+h_part[i-1]->v;
    }
    return arr;
}

__global__ void kernel_BWD(Bwd_d* d_graph){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < d_graph->v){

    }
}
int* findBWD(Bwd_d** d_part, Bwd** h_part, int* h_pivots, int* number_of_blocks, int* number_of_thread_per_block){
    int* startPoints = startPointForCopy(h_part);
    for(int i=0;i<NGPUS;i++){
        hipSetDevice(i);
        hipMemcpy((d_part[i]->d_bwd_v), (h_pivots+startPoints[i]), h_part[i]->v*sizeof(int), hipMemcpyHostToDevice);
    }

    for(int i=0;i<NGPUS;i++){
        hipSetDevice(i);
        kernel_BWD<<<number_of_blocks[i],number_of_thread_per_block[i]>>>(d_part[i]);
    }
}
