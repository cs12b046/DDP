#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <string>
#include <unistd.h>
#include <time.h>
#include "Edge.h"
#include "kernel.c"
#include "convert.c"

int main(int argc, char const *argv[]) {
    //int* h_e1;
    //int* h_e2;
    //int* d_e1;
    //int* d_e2;
    int v,e;
    int* h_terminate_color;
    int* d_terminate_color;
    int* h_graph;
    int* d_graph;
    int* h_allowed_v;
    int* d_allowed_v;
    int* h_v;
    int* d_v;
    int* d_v_prev;
    int* h_out_colors;
    int* d_out_colors;
    int* d_terminate_BWD;
    int* h_terminate_BWD;
    int* d_terminate_trim;
    int* h_terminate_trim;
    int* d_prev_BWD;
    int* h_leftout;
    int* d_leftout;
    int* h_algo_out;
    int* d_algo_out;
    int* d_count;
    int* h_count;
    int* old;
    int number_of_iteration = 0;

    /* Code for take input graph */
    if(argc < 3){
        std::cout<<"Error : Invalid Arguents\n";
        return 0;
    }
    FILE* fp;
    fp = fopen(argv[1],"r");
    fscanf(fp,"%d %d",&v, &e);
    hipMalloc((void **)&old,v*sizeof(int));

    /* Allocates memory for edges at host and at device*/
    //h_e1 = (int *)malloc(e*sizeof(int));
    //h_e2 = (int *)malloc(e*sizeof(int));
    //hipMalloc((void **)&d_e1,e*sizeof(int));
    //hipMalloc((void **)&d_e2,e*sizeof(int));

    /* Taking input from file */
    Edge* edgeList;
    edgeList = (Edge *)malloc(e*sizeof(Edge));
    for(int i=0;i<e;i++){
        Edge temp_edge;
        fscanf(fp,"%d %d",&temp_edge.src,&temp_edge.dest);
	edgeList[i] = temp_edge;
    }
    std::cout << "/* Reading done */" << std::endl;
    /* Convert Graph in CSR */
    GraphCSR* h_g = convertToCSR(v,e,edgeList);
    int* d_vlist;
    int* d_elist;
    hipMalloc((void **)&d_vlist, (v+1)*sizeof(int));
    hipMemcpy(d_vlist,h_g->vlist,(v+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMalloc((void **)&d_elist, (e)*sizeof(int));
    hipMemcpy(d_elist,h_g->elist,(e)*sizeof(int),hipMemcpyHostToDevice);
    std::cout << "/* Converted into CSR */" << std::endl;
    /* Convert graph_prime */
    GraphCSR* h_g_prime = convertToCSRPrime(v,e, edgeList);
    int* d_vlist_prime;
    int* d_elist_prime;
    hipMalloc((void **)&d_vlist_prime, (v+1)*sizeof(int));
    hipMemcpy(d_vlist_prime,h_g_prime->vlist,(v+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMalloc((void **)&d_elist_prime, (e)*sizeof(int));
    hipMemcpy(d_elist_prime,h_g_prime->elist,(e)*sizeof(int),hipMemcpyHostToDevice);
    std::cout << "/* Converted into CSR (G_Prime) */" << std::endl;
    //hipMemcpy(graph,d_g,sizeof(GraphCSR),hipMemcpyDeviceToHost);
    /* This is not the part of program this is part of debugging please remove it after debugging */
    // std::cout<<"Graph in CSR format"<<std::endl;
    // // GraphCSR g = convertToCSR(v,e,h_e1,h_e2);
    // for(int i=0;i<v+1;i++){
    //     std::cout<<graph->vlist[i]<<" ";
    // }
    // std::cout<<std::endl;
    // for(int i = 0;i<e;i++){
    //     std::cout<<graph->elist[i]<<" ";
    // }
    // std::cout<<std::endl;
    // std::cout<<"Graph ends Here"<<std::endl;
    /* Till here */

    /* Copying edge List to device */
    //hipMemcpy(d_e1,h_e1,e*sizeof(int),hipMemcpyHostToDevice);
    //hipMemcpy(d_e2,h_e2,e*sizeof(int),hipMemcpyHostToDevice);

    /* Setup for algorithm */
    h_terminate_color = (int*) malloc(sizeof(int));
    h_allowed_v = (int*) malloc(v*sizeof(int));
    h_graph = (int* )malloc(2*sizeof(int));
    h_v = (int* )malloc(v*sizeof(int));
    h_out_colors = (int* )malloc(v*sizeof(int));
    h_terminate_BWD = (int *)malloc(sizeof(int));
    h_terminate_trim = (int *)malloc(sizeof(int));
    h_leftout = (int*) malloc(sizeof(int));
    h_algo_out = (int* )malloc(v*sizeof(int));
    h_count = (int*) malloc(sizeof(int));

    hipMalloc((void **)&d_v,v*sizeof(int));
    hipMalloc((void **)&d_graph,2*sizeof(int));
    hipMalloc((void **)&d_allowed_v,v*sizeof(int));
    hipMalloc((void **)&d_terminate_color,sizeof(int));
    hipMalloc((void **)&d_terminate_trim,sizeof(int));
    hipMalloc((void **)&d_terminate_BWD,sizeof(int));
    hipMalloc((void **)&d_v_prev,v*sizeof(int));
    hipMalloc((void **)&d_out_colors,v*sizeof(int));
    hipMalloc((void **)&d_prev_BWD,v*sizeof(int));
    hipMalloc((void **)&d_leftout,sizeof(int));
    hipMalloc((void **)&d_algo_out,v*sizeof(int));
    hipMalloc((void **)&d_count,sizeof(int));

    h_graph[0] = v;
    h_graph[1] = e;
    /* Start timer here */
    clock_t tStart = clock();
    hipMemcpy(d_graph,h_graph,2*sizeof(int), hipMemcpyHostToDevice);

    /*  Finding numbers of blocks for vertex parallelism */
    int number_of_blocks = ceil((v*1.0)/512.0);
    int number_of_thread_per_block = ceil(v/number_of_blocks)+1;
    int number_of_blocks_e = ceil((e*1.0)/512.0);
    int number_of_thread_per_block_e = ceil(e/number_of_blocks)+1;

    /* Init d_allowed_v and h_allowed_v */
    kernel_Init_Allowed_Vert<<<number_of_blocks,number_of_thread_per_block>>>(d_allowed_v,d_graph);
    hipMemcpy(h_allowed_v,d_allowed_v,v*sizeof(int),hipMemcpyDeviceToHost);

    h_leftout[0] = v;
    hipMemcpy(d_leftout,h_leftout,sizeof(int),hipMemcpyHostToDevice);

    // /* Just for debugging */
    // kernel_COLORING<<<number_of_blocks,number_of_thread_per_block>>>(d_graph,d_vlist,d_elist,d_v,d_allowed_v);
    // hipMemcpy(h_v,d_v,v*sizeof(int),hipMemcpyDeviceToHost);
    // for(int i=0;i<v;i++){
    //         std::cout<<h_v[i]<< " ";
    // }
    // std::cout<<std::endl;
    // return 0;
    // /* Till here */
    /* Init old values*/
    //hipMemset(old,-1,v*sizeof(int));
   // int* h_old = (int *)malloc(v*sizeof(int));
    //hipMemcpy(h_old,old,v*sizeof(int),hipMemcpyDeviceToHost);
    //std::cout<<"old[0] "<<old[0]<<std::endl;
    //sleep(1);
    //boost::this_thread::sleep( boost::posix_time::seconds(1) );
    //kernel_init_old<<<1,1>>>(d_graph);
    while(h_leftout[0] > 0){
        //sleep(3);
	number_of_iteration++;
        kernel_Init_Color<<<number_of_blocks,number_of_thread_per_block>>>(d_v,d_graph,d_allowed_v);
        /* code for trimming */
        h_terminate_trim[0] = 0;
        while(h_terminate_trim[0] == 0){
            h_terminate_trim[0] = 1;
            hipMemcpy(d_terminate_trim,h_terminate_trim,sizeof(int),hipMemcpyHostToDevice);
            kernel_trim<<<number_of_blocks,number_of_thread_per_block>>>(d_v,d_vlist,d_elist,d_graph,d_allowed_v,d_terminate_trim, d_leftout);
            kernel_trim<<<number_of_blocks,number_of_thread_per_block>>>(d_v,d_vlist_prime,d_elist_prime,d_graph,d_allowed_v,d_terminate_trim, d_leftout);
            hipMemcpy(h_terminate_trim,d_terminate_trim,sizeof(int),hipMemcpyDeviceToHost);
        }
	/*std::cout<<"TRIM END"<<std::endl;*/
        //std::cout << "/* END OF TRIM */" << std::endl;
        //kernel_Init_Color_Prev<<<number_of_blocks,number_of_thread_per_block>>>(d_v_prev,d_graph,d_allowed_v);
        /* Propogating Colors */
        h_terminate_color[0] = 0;
        while(h_terminate_color[0] == 0){
            h_terminate_color[0] = 1;
            hipMemcpy(d_terminate_color,h_terminate_color,sizeof(int),hipMemcpyHostToDevice);
            //kernel_COLORING<<<number_of_blocks,number_of_thread_per_block>>>(d_e1,d_e2,d_v,d_allowed_v,d_graph);
            kernel_COLORING<<<number_of_blocks,number_of_thread_per_block>>>(d_graph,d_vlist,d_elist,d_v,d_allowed_v,d_terminate_color,old);
            //terminate<<<number_of_blocks,number_of_thread_per_block>>>(d_v_prev,d_v, d_terminate_color,d_graph,d_allowed_v);
            //hipMemcpy(d_v_prev,d_v,v*sizeof(int),hipMemcpyDeviceToDevice);
            hipMemcpy(h_terminate_color,d_terminate_color,sizeof(int),hipMemcpyDeviceToHost);
            // if(atoi(argv[2]) > 0)
            //hipMemcpy(h_v,d_v,v*sizeof(int),hipMemcpyDeviceToHost);
        }
        // std::cout << "/* At iteration " <<number_of_iteration<<" */"<< std::endl;
        // for(int iter=0;iter<v;iter++){
        //     std::cout <<iter<<"-->"<<h_v[iter]<<std::endl;
        // }
        //std::cout << "/* END OF COLOR Propogating */" << std::endl;
        /* Find vertex with the original colors */
        kernel_vertex_with_org_color<<<number_of_blocks,number_of_thread_per_block>>>(d_v,d_allowed_v,d_graph,d_out_colors);
        hipMemcpy(h_out_colors,d_out_colors,v*sizeof(int),hipMemcpyDeviceToHost);

        // for(int i=0;i<v;i++){
        //         std::cout<<h_out_colors[i]<< " ";
        // }
        // std::cout<<std::endl;

        /* this kernel for init d_prev_BWD */
        //kernel_Init<<<number_of_blocks,number_of_thread_per_block>>>(d_prev_BWD,d_graph,d_allowed_v);

        /* Code for backward reach */
        h_terminate_BWD[0] = 0;
        while(h_terminate_BWD[0] == 0){
            h_terminate_BWD[0] = 1;
            hipMemcpy(d_terminate_BWD,h_terminate_BWD,sizeof(int),hipMemcpyHostToDevice);
            //kernel_BWD<<<number_of_blocks,number_of_thread_per_block>>>(d_graph,d_e2,d_e1,d_allowed_v,d_out_colors, d_v);
            kernel_BWD<<<number_of_blocks,number_of_thread_per_block>>>(d_graph,d_vlist_prime,d_elist_prime,d_allowed_v,d_out_colors,d_v,d_terminate_BWD,old);
            //terminate<<<number_of_blocks,number_of_thread_per_block>>>(d_prev_BWD,d_out_colors,d_terminate_BWD,d_graph,d_allowed_v);
            //hipMemcpy(h_out_colors,d_out_colors,v*sizeof(int),hipMemcpyDeviceToHost);
            hipMemcpy(h_terminate_BWD,d_terminate_BWD,sizeof(int),hipMemcpyDeviceToHost);
            //hipMemcpy(d_prev_BWD,d_out_colors,v*sizeof(int),hipMemcpyDeviceToDevice);
            // for(int i=0;i<v;i++){
            //         std::cout<<h_out_colors[i]<< " ";
            // }
            // std::cout<<std::endl;
        }
        //std::cout << "/* END OF BWD */" << std::endl;
        kernel_change_allowed<<<number_of_blocks,number_of_thread_per_block>>>(d_allowed_v,d_out_colors,d_graph,d_leftout);
        hipMemcpy(h_leftout,d_leftout,sizeof(int),hipMemcpyDeviceToHost);
        // hipMemcpy(h_v,d_v,v*sizeof(int),hipMemcpyDeviceToHost);
        // for(int i=0;i<v;i++){
        //         std::cout<<h_v[i]<< " ";
        // }
        //std::cout << "/* END OF leftout */ " << h_leftout[0]<< std::endl;
	//std::cout<<h_leftout[0]<<std::endl;
    }

    /* Code for printing final color */
    int choice = atoi(argv[2]);
    if(choice >= 1){
        hipMemcpy(h_v,d_v,v*sizeof(int),hipMemcpyDeviceToHost);
        std::cout<<"============COLORS=============="<<std::endl;
        for(int i=0;i<v;i++){
            std::cout<<i<<" --> "<<h_v[i]<<std::endl;
        }
        std::cout<<"=============SCC================"<<std::endl;
        /* Code for extracting SCC from colors*/
        for(int i=0;i<v;i++){
            h_count[0] = 0;
            kernel_Init_SCC<<<number_of_blocks,number_of_thread_per_block>>>(d_algo_out,d_graph);
            hipMemcpy(d_count,h_count,sizeof(int),hipMemcpyHostToDevice);
            kernel_SCC<<<number_of_blocks,number_of_thread_per_block>>>(d_v,i,d_algo_out,d_graph,d_count);
            hipMemcpy(h_algo_out,d_algo_out,v*sizeof(int),hipMemcpyDeviceToHost);
            hipMemcpy(h_count,d_count,sizeof(int),hipMemcpyDeviceToHost);
            for(int j=0;j<v;j++){
                  if(h_algo_out[j] > 0)
                    std::cout<<j<<" ";
            }
            if(h_count[0] > 0)
                std::cout<<std::endl;
        }
        std::cout<<"================================"<<std::endl;
    }
    /* Printing number of cluster and time taken by the progam */
    clock_t tEnd = clock();
    int* d_total_scc;
    int* h_total_scc = (int*)malloc(sizeof(int));
    hipMalloc((void**)&d_total_scc,sizeof(int));
    h_total_scc[0] = 0;
    hipMemcpy(d_total_scc,h_total_scc,sizeof(int),hipMemcpyHostToDevice);
    kernel_Total_SCC<<<number_of_blocks,number_of_thread_per_block>>>(d_graph,d_v, d_total_scc);
    hipMemcpy(h_total_scc,d_total_scc,sizeof(int),hipMemcpyDeviceToHost);
    std::cout<<"|V| = "<<v<<" |e| = "<<e<<std::endl;
    std::cout<<"Total SCC are -> "<<h_total_scc[0]<<std::endl;
    printf("Time taken: %.2fs\n", (double)(tEnd - tStart)/CLOCKS_PER_SEC);
    printf("/* Number of iteration %d */\n",number_of_iteration);
    /* Freeing all the space used */
    //free(h_e1);
    //free(h_e2);
    //hipFree(d_e1);
    //hipFree(d_e2);
    free(h_terminate_color);
    hipFree(d_terminate_color);
    free(h_graph);
    hipFree(d_graph);
    free(h_allowed_v);
    hipFree(d_allowed_v);
    free(h_v);
    hipFree(d_v);
    hipFree(d_v_prev);
    free(h_out_colors);
    hipFree(d_out_colors);
    hipFree(d_terminate_BWD);
    free(h_terminate_BWD);
    hipFree(d_prev_BWD);
    free(h_leftout);
    hipFree(d_leftout);
    free(h_algo_out);
    hipFree(d_algo_out);
    hipFree(d_count);
    free(h_count);

    return 0;
}
